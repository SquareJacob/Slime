#include "hip/hip_runtime.h"
#include <SDL.h>
#include <SDL_image.h>
#include <SDL_ttf.h>
#include <SDL_mixer.h>
#include <iostream>
#include <stdlib.h>  
#include <crtdbg.h>   //for malloc and free
#include <set>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#define _CRTDBG_MAP_ALLOC
#ifdef _DEBUG
#define new new( _NORMAL_BLOCK, __FILE__, __LINE__)
#endif

SDL_Window* window;
SDL_Renderer* renderer;
bool running;
SDL_Event event;
std::set<std::string> keys;
std::set<std::string> currentKeys;
int mouseX = 0;
int mouseY = 0;
int mouseDeltaX = 0;
int mouseDeltaY = 0;
int mouseScroll = 0;
std::set<int> buttons;
std::set<int> currentButtons;
const int WIDTH = 800;
const int HEIGHT = 600;

__global__ void initCurand(unsigned int seed, hiprandState* state) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

const double TRAILDECAY = 0.01;
const double DIFFUSION = 20.0; //inverse
double pheremones[HEIGHT * WIDTH] = { 0.0 };
double newP[HEIGHT * WIDTH];
double *d_newP, *d_pheremones;
size_t s_pheremones = sizeof(double) * static_cast<size_t>(WIDTH) * static_cast<size_t>(HEIGHT);
__global__ void diffuseTrail(double* pheremones, double* newP, double DIFFUSION) {
	int i = blockIdx.x;
	int j = threadIdx.x;
	int x, y;
	double sum = 0.0;
	for (int k = -1; k < 2; k++) {
		for (int l = -1; l < 2; l++) {
			x = i + k;
			y = j + l;
			if (k == 0 && l == 0) {
				sum += pheremones[y * WIDTH + x] * DIFFUSION;
			}
			else if (-1 < x && x < WIDTH && -1 < y && y < HEIGHT) {
				sum += pheremones[y * WIDTH + x];
			}
		}
	}
	newP[j * WIDTH + i] = sum / (8.0 + DIFFUSION);
	//newP[j * WIDTH + i] = 1.0;
}
__global__ void copyTrail(double* pheremones, double* newP) {
	pheremones[threadIdx.x * WIDTH + blockIdx.x] = newP[threadIdx.x * WIDTH + blockIdx.x];
	//pheremones[threadIdx.x * WIDTH + blockIdx.x] = 1.0;
}

class Cell {
public:
	double x = 0.0, y = 0.0, angle = 0.0;
	__device__ bool move(double speed) {
		double deltaX = speed * cos(angle);
		double deltaY = speed * sin(angle);
		if (0.0 < x + deltaX && x + deltaX < WIDTH && 0.0 < y + deltaY && y + deltaY < HEIGHT) {
			x += deltaX;
			y += deltaY;
			return true;
		}
		else {
			return false;
		}
	}
	__device__ void sense(hiprandState* state, double sensorDistance, double sensorAngle, double rotateAmount, double* pheremones) {
		double frontSensor = pheremones[static_cast<int>(y + sensorDistance * sin(angle) + 0.5) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle) + 0.5)];
		double leftSensor = pheremones[static_cast<int>(y + sensorDistance * sin(angle + sensorAngle) + 0.5) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle + sensorAngle) + 0.5)];
		double rightSensor = pheremones[static_cast<int>(y + sensorDistance * sin(angle - sensorAngle) + 0.5) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle - sensorAngle) + 0.5)];
		if (frontSensor > leftSensor && frontSensor > rightSensor) {
			return;
		}
		else if (frontSensor < leftSensor && frontSensor < rightSensor) {
			angle += static_cast<float>(2 * hiprand(state) % 2 - 1) * rotateAmount;
		}
		else if (rightSensor > leftSensor) {
			angle -= rotateAmount;
		}
		else if (rightSensor < leftSensor) {
			angle += rotateAmount;
		}
	}
	void draw() {
		SDL_RenderDrawPoint(renderer, static_cast<int>(x), static_cast<int>(y));
	}
	__device__ void trail(double* pheremones) {
		pheremones[static_cast<int>(y) * WIDTH + static_cast<int>(x)] = 1.0;
	}
};
double speed = 1.0;
double sensorDistance = 10.0;
double sensorAngle = M_PI / 4;
double rotateAmount = M_PI / 16;
const int CELLCOUNT = 10000; //KEEP SQUARE AND LESS THAN 1024^2
const int CELLCOUNTSQRT = 100; //KEEP AS SQRT OF CELLCOUNT
Cell cells[CELLCOUNT];
Cell* d_cells;
size_t s_cells = sizeof(Cell) * static_cast<size_t>(CELLCOUNT);

__global__ void moveCell(Cell* cells, hiprandState* state, double speed, double* pheremones) {
	int i = CELLCOUNTSQRT * threadIdx.x + blockIdx.x;
	if (cells[i].move(speed)) {
		cells[i].trail(pheremones);
	}
	else {
		cells[i].angle = hiprand_uniform(state) * 2.0 * M_PI;
	}
}
__global__ void sense(Cell* cells, hiprandState* state, double sensorDistance, double sensorAngle, double rotateAmount, double* pheremones) {
	cells[CELLCOUNTSQRT * threadIdx.x + blockIdx.x].sense(state, sensorDistance, sensorAngle, rotateAmount, pheremones);
}

void debug(int line, std::string file) {
	std::cout << "Line " << line << " in file " << file << ": " << SDL_GetError() << std::endl;
}

double random() {
	return static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
}

__device__ Uint32 red = 0x01000000, blue = 0x00010000, green = 0x00000100;
__global__ void pixelize(double* pheremones, Uint32* pixel_ptr, double TRAILDECAY) {
	double* p = &pheremones[threadIdx.x * WIDTH + blockIdx.x];
	if (*p > 0.0) {
		*p = *p - TRAILDECAY;
		if (*p < 0.0) {
			*p = 0.0;
		}
	}
	pixel_ptr[threadIdx.x * WIDTH + blockIdx.x] = static_cast<Uint32>(*p * 255) * (red + green + blue) + 255;
}
Uint32* pixel_ptr, *d_pixel_ptr, *pixel_ptrA;
size_t s_pixel_ptr = sizeof(Uint32) * static_cast<size_t>(WIDTH * HEIGHT);

Uint32 frameStart, calcStart, drawStart;
int frameTime = 0;
bool timing = true;
int main(int argc, char* argv[]) {
	srand(time(0));
	if (SDL_Init(SDL_INIT_EVERYTHING) == 0 && TTF_Init() == 0 && Mix_OpenAudio(44100, MIX_DEFAULT_FORMAT, 2, 2048) == 0) {
		//Setup
		window = SDL_CreateWindow("Window", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, WIDTH, HEIGHT, 0);
		if (window == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}

		renderer = SDL_CreateRenderer(window, -1, 0);
		if (renderer == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}

		hipSetDevice(0);
		hiprandState* d_state;
		hipMalloc(&d_state, sizeof(hiprandState));
		initCurand << <1, 1 >> > (time(0), d_state);
		hipMalloc((void**)&d_pheremones, s_pheremones);
		hipMalloc((void**)&d_newP, s_pheremones);
		hipMalloc((void**)&d_cells, s_cells);
		hipMalloc((void**)&d_pixel_ptr, s_pixel_ptr);

		SDL_Texture* texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGBA8888,
			SDL_TEXTUREACCESS_STREAMING, WIDTH, HEIGHT);
		SDL_Texture* textureA = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGBA8888,
			SDL_TEXTUREACCESS_STREAMING, WIDTH, HEIGHT);
		void* txtPixels;
		int pitch;
		SDL_PixelFormat* format = SDL_AllocFormat(SDL_PIXELFORMAT_RGBA8888);

		double angle;
		double radius;
		for (int i = 0; i < CELLCOUNT; i++) {
			angle = random() * 2.0 * M_PI;
			radius = std::min(HEIGHT, WIDTH) * random() / 2;
			cells[i].angle = angle;
			cells[i].x = static_cast<float>(WIDTH) / 2.0 - radius * cos(angle);
			cells[i].y = static_cast<float>(HEIGHT) / 2.0 - radius * sin(angle);
		}

		//Main loop
		running = true;
		while (running) {
			//handle events
			frameStart = SDL_GetTicks();
			for (std::string i : keys) {
				currentKeys.erase(i); //make sure only newly pressed keys are in currentKeys
			}
			for (int i : buttons) {
				currentButtons.erase(i); //make sure only newly pressed buttons are in currentButtons
			}
			mouseScroll = 0;
			while (SDL_PollEvent(&event)) {
				switch (event.type) {
				case SDL_QUIT:
					running = false;
					break;
				case SDL_KEYDOWN:
					if (!keys.contains(std::string(SDL_GetKeyName(event.key.keysym.sym)))) {
						currentKeys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym)));
					}
					keys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym))); //add keydown to keys set
					break;
				case SDL_KEYUP:
					keys.erase(std::string(SDL_GetKeyName(event.key.keysym.sym))); //remove keyup from keys set
					break;
				case SDL_MOUSEMOTION:
					mouseX = event.motion.x;
					mouseY = event.motion.y;
					mouseDeltaX = event.motion.xrel;
					mouseDeltaY = event.motion.yrel;
					break;
				case SDL_MOUSEBUTTONDOWN:
					if (!buttons.contains(event.button.button)) {
						currentButtons.insert(event.button.button);
					}
					buttons.insert(event.button.button);
					break;
				case SDL_MOUSEBUTTONUP:
					buttons.erase(event.button.button);
					break;
				case SDL_MOUSEWHEEL:
					mouseScroll = event.wheel.y;
					break;
				}
			}

			calcStart = SDL_GetTicks();
			hipMemcpy(d_pheremones, pheremones, s_pheremones, hipMemcpyHostToDevice);
			hipMemcpy(d_newP, newP, s_pheremones, hipMemcpyHostToDevice);
			hipMemcpy(d_cells, cells, s_cells, hipMemcpyHostToDevice);
			diffuseTrail << <WIDTH, HEIGHT >> > (d_pheremones, d_newP, DIFFUSION);
			copyTrail << <WIDTH, HEIGHT >> > (d_pheremones, d_newP);
			moveCell << <CELLCOUNTSQRT, CELLCOUNTSQRT >> > (d_cells, d_state, speed, d_pheremones);
			sense << <CELLCOUNTSQRT, CELLCOUNTSQRT >> > (d_cells, d_state, sensorDistance, sensorAngle, rotateAmount, d_pheremones);
			hipDeviceSynchronize();
			hipMemcpy(pheremones, d_pheremones, s_pheremones, hipMemcpyDeviceToHost);
			hipMemcpy(newP, d_newP, s_pheremones, hipMemcpyDeviceToHost);
			hipMemcpy(cells, d_cells, s_cells, hipMemcpyDeviceToHost);
			if (timing) {
				std::cout << "calc time: " << SDL_GetTicks() - calcStart;
			}

			drawStart = SDL_GetTicks();
			SDL_SetRenderDrawColor(renderer, 0, 0, 0, 255);
			SDL_RenderClear(renderer);
			SDL_LockTexture(texture, NULL, &txtPixels, &pitch);
			pixel_ptr = (Uint32*)txtPixels;

			hipMemcpy(d_pixel_ptr, pixel_ptr, s_pixel_ptr, hipMemcpyHostToDevice);
			pixelize << <WIDTH, HEIGHT >> > (d_pheremones, d_pixel_ptr, TRAILDECAY);
			hipDeviceSynchronize();
			hipMemcpy(pixel_ptr, d_pixel_ptr, s_pixel_ptr, hipMemcpyDeviceToHost);

			SDL_UnlockTexture(texture);
			//SDL_RenderCopy(renderer, texture, NULL, NULL);
			SDL_SetRenderDrawColor(renderer, 255, 255, 255, 255);
			for (int i = 0; i < CELLCOUNT; i++) {
				cells[i].draw();
			}
			SDL_RenderPresent(renderer);
			if (timing) {
				std::cout << " draw time: " << SDL_GetTicks() - drawStart;
			}
			frameTime = SDL_GetTicks() - frameStart;
			if (timing) {
				std::cout << " total time: " << frameTime << std::endl;
			}
		}

		//Clean up
		SDL_FreeFormat(format);
		SDL_DestroyTexture(texture);
		hipFree(d_pheremones);
		hipFree(d_newP);
		if (window) {
			SDL_DestroyWindow(window);
		}
		if (renderer) {
			SDL_DestroyRenderer(renderer);
		}
		TTF_Quit();
		Mix_Quit();
		IMG_Quit();
		SDL_Quit();
		return 0;
	}
	else {
		return 0;
	}
}